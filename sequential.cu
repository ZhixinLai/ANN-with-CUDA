/********************************************************************

sequential.cu the sequential version of NN 

Input: 
/usr/local/cuda-10.1/bin/nvcc -arch=compute_52 -o sequential.out sequential.cu
./sequential.out block_size activationtype     // block_size = 0; activationtype=1 means sigomid and 2 means ReLU

Output: 
elapsed_time - the elapsed time to perform the multiplication.
accuracy on training set and test set.

********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#include <iostream>  
#include <string>  
#include <vector>  
#include <fstream>  
#include <sstream> 
#include<random> 
using namespace std; 


#define X_trn(x, y) X_trn[x * size_train + y] // 196 * 964
#define X_tst(x, y) X_tst[x * size_test + y]  // 196 * 414
#define Y_trn(x, y) Y_trn[x * size_train + y] // 1   * 964
#define Y_tst(x, y) Y_tst[x * size_test + y]  // 1   * 414
#define X(x, y) X[x * size_batch + y]  // 196 * 964
#define Y(x, y) Y[x * size_batch + y]  // 1   * 414


#define W1(x, y) W1[x * size_input + y]       // 20 * 196
#define b1(x, y) b1[x * 1 + y]                // 20 * 1
#define W2(x, y) W2[x * size_hidden + y]      // 2  * 20
#define b2(x, y) b2[x * 1 + y]                // 2  * 1

#define dW1(x, y) dW1[x * size_input + y]     // 20 * 196
#define db1(x, y) db1[x * 1 + y]              // 20 * 1
#define dW2(x, y) dW2[x * size_hidden + y]    // 2  * 20
#define db2(x, y) db2[x * 1 + y]              // 2  * 1

#define Z1(x, y) Z1[x * size_batch + y]       // 20 * 964
#define A1(x, y) A1[x * size_batch + y]       // 20 * 964
#define Z2(x, y) Z2[x * size_batch + y]       // 2  * 964
#define A2(x, y) A2[x * size_batch + y]       // 2  * 964

#define dZ1(x, y) dZ1[x * size_batch + y]     // 20 * 964
#define dA1(x, y) dA1[x * size_batch + y]     // 20 * 964
#define dZ2(x, y) dZ2[x * size_batch + y]     // 2  * 964
#define dA2(x, y) dA2[x * size_batch + y]     // 2  * 964


#define dev_X_trn(x, y) dev_X_trn[x * size_train + y] // 196 * 964
#define dev_X_tst(x, y) dev_X_tst[x * size_test + y]  // 196 * 414
#define dev_Y_trn(x, y) dev_Y_trn[x * size_train + y] // 1   * 964
#define dev_Y_tst(x, y) dev_Y_tst[x * size_test + y]  // 1   * 414
#define dev_X(x, y) dev_X[x * size_batch + y] // 196 * 964
#define dev_Y(x, y) dev_Y[x * size_batch + y]  // 1   * 414


#define dev_W1(x, y) dev_W1[x * size_input + y]       // 20 * 196
#define dev_b1(x, y) dev_b1[x * 1 + y]                // 20 * 1
#define dev_W2(x, y) dev_W2[x * size_hidden + y]      // 2  * 20
#define dev_b2(x, y) dev_b2[x * 1 + y]                // 2  * 1

#define dev_dW1(x, y) dev_dW1[x * size_input + y]     // 20 * 196
#define dev_db1(x, y) dev_db1[x * 1 + y]              // 20 * 1
#define dev_dW2(x, y) dev_dW2[x * size_hidden + y]    // 2  * 20
#define dev_db2(x, y) dev_db2[x * 1 + y]              // 2  * 1

#define dev_Z1(x, y) dev_Z1[x * size_batch + y]       // 20 * 964
#define dev_A1(x, y) dev_A1[x * size_batch + y]       // 20 * 964
#define dev_Z2(x, y) dev_Z2[x * size_batch + y]       // 2  * 964
#define dev_A2(x, y) dev_A2[x * size_batch + y]       // 2  * 964

#define dev_dZ1(x, y) dev_dZ1[x * size_batch + y]     // 20 * 964
#define dev_dA1(x, y) dev_dA1[x * size_batch + y]     // 20 * 964
#define dev_dZ2(x, y) dev_dZ2[x * size_batch + y]     // 2  * 964
#define dev_dA2(x, y) dev_dA2[x * size_batch + y]     // 2  * 964

#define max_index(x, y) max_index[y] // 1  * 964

int size_train  = 964;
int size_test   = 414;
int size_batch  = 0;

int size_input  = 196;
int size_hidden = 20;
int size_output = 2;

int size_X_trn = 196*964;
int size_Y_trn = 1*964;
int size_X_tst = 196*414;
int size_Y_tst = 1*414;
int size_Xbatch = 0;
int size_Ybatch = 0;


int size_W1 = size_hidden*size_input;
int size_b1 = size_hidden*1;
int size_W2 = size_output*size_hidden;
int size_b2 = size_output*1;

int size_dW1 = size_hidden*size_input;
int size_db1 = size_hidden*1;
int size_dW2 = size_output*size_hidden;
int size_db2 = size_output*1;

#define size_Z1 size_hidden*size_batch
#define size_A1 size_hidden*size_batch
#define size_Z2 size_output*size_batch
#define size_A2 size_output*size_batch

#define size_dZ1 size_hidden*size_batch
#define size_dA1 size_hidden*size_batch
#define size_dZ2 size_output*size_batch
#define size_dA2 size_output*size_batch

#define size_max_index 1*size_batch

double *X_trn, *X_tst;
int *Y_trn, *Y_tst;
double *W1, *b1, *W2, *b2;
double *dW1, *db1, *dW2, *db2;
double *Z1, *A1, *Z2, *A2;
double *dZ1, *dA1, *dZ2, *dA2;
int *max_index;



void HiddenLayer(double* dev_X, double* dev_W1, double* dev_b1, double* dev_A1, double* dev_Z1, int size_input, int size_batch, int acti_type, int max_row, int max_col)

{

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) {
      double partial = 0.0;
    	for (int k = 0; k < size_input; k++){
    		partial += dev_W1(i,k) * dev_X(k,j);
      }
    	dev_Z1(i,j) = partial + dev_b1(i,0);
	    // Sigmoid
    	if (acti_type == 1)
    		dev_A1(i,j) = 1 / (1 + exp(0 - dev_Z1(i,j)));
    	// ReLU
    	if (acti_type == 2) {
    		if (dev_Z1(i,j) < 0)
    			dev_A1(i,j) = 0;
    		if (dev_Z1(i,j) >= 0)
    			dev_A1(i,j) = dev_Z1(i,j);
    	}
    }
  }
}

void OutputLayer(double* dev_A1, double* dev_W2, double* dev_b2, double* dev_Z2, int size_hidden, int size_batch, int max_row, int max_col)

{

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) {
	    double partial = 0.0;
    	for (int k = 0; k < size_hidden; k++){
    		partial += dev_W2(i,k) * dev_A1(k,j);
      }
    	dev_Z2(i,j) = partial + dev_b2(i,0);
    }
  }
 
}

void Softmax(double* Z2, int row, int col, double* A2, int* max_index)
{

  int c, r;  
	double max = 0, sum = 0;
	for (c = 0; c < col; c++) {
    max = Z2(0, c);
    max_index[c] = 1;    
		for (r = 1; r < row; r++) {   
			if (Z2(r, c) > max){      
				max = Z2(r, c);        
        max_index[c] = 0;        
      }
		}
		sum = 0;
		for (r = 0; r < row; r++)
			sum += exp(Z2(r, c));
		for (r = 0; r < row; r++)
			A2(r, c) = exp(Z2(r, c)) / sum;
  }
  return;

}


double cross_entropy_loss(int* Y, double* A2, int col) 
{
  
  int c;
  double loss = 0;
  for(c = 0; c < col; c++) {
    loss += -log(A2(0, c)) * Y(0, c) - log(A2(1, c)) * (1-Y(0, c));
  }
  return loss/col;
  
}

/* init Z and A in the host */
void initialize_ZA(int size_batch) {

  Z1 = (double *) malloc(size_Z1*sizeof(double));   // 20*964
  A1 = (double *) malloc(size_A1*sizeof(double));   // 20*964
  Z2 = (double *) malloc(size_Z2*sizeof(double));   // 2*964
  A2 = (double *) malloc(size_A2*sizeof(double));   // 2*964

  dZ1 = (double *) malloc(size_dZ1*sizeof(double));  // 20*964
  dA1 = (double *) malloc(size_dA1*sizeof(double));  // 20*964
  dZ2 = (double *) malloc(size_dZ2*sizeof(double));  // 2*964
  dA2 = (double *) malloc(size_dA2*sizeof(double));  // 2*964
  
  max_index = (int *) malloc(size_max_index*sizeof(int));             // 1*964
    
  memset (Z1,0,  size_Z1);
  memset (A1,0,  size_A1);
  memset (Z2,0,  size_Z2);
  memset (A2,0,  size_A2);
  
  memset (dZ1,0, size_dZ1);
  memset (dA1,0, size_dA1);
  memset (dZ2,0, size_dZ2);
  memset (dA2,0, size_dA2);
  
  memset (max_index,0,size_max_index);

}

void forward(double* X, int* Y, string type, int acti_type,  int block_size){

  if(type == "train"){
    size_batch  = size_train;
    size_Xbatch = size_X_trn;
    size_Ybatch = size_Y_trn;        
  }
  else{
    size_batch = size_test;
    size_Xbatch = size_X_tst;
    size_Ybatch = size_Y_tst;    
  }

  // init Z and A in the host
  initialize_ZA(size_batch);

  // hidden layer and activation function to get Z1 and A1
  HiddenLayer(X, W1, b1, A1, Z1, size_input, size_batch, acti_type, size_hidden, size_batch); 

  // output layer to get Z2
  OutputLayer(A1, W2, b2, Z2, size_hidden, size_batch, size_output, size_batch);
 
  // softmax layer to get A2
  Softmax(Z2, size_output, size_batch, A2, max_index);

}


void Back_dZ2 (double* dev_A2, int* dev_Y_trn, double* dev_dZ2, int size_train, int size_batch, int max_row, int max_col)

{

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) {
    	// int c = threadIdx.x; // column of Z2
      dev_dZ2(0, j) = (dev_A2(0, j) - dev_Y_trn(0, j)) / size_train;
      dev_dZ2(1, j) = (dev_Y_trn(0, j) - dev_A2(0, j)) / size_train;
     }
  }

}

// dW1(20*196) = dZ1(20*964) * X(196*964)
// dW2(2*20) = dZ2(2*964) * A1(20*964)
void Back_dW (double* dev_A, double* dev_dZ, double* dev_dW, int size_batch, int W_col, int max_row, int max_col)

{  

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) {
      double tmp = 0.0;
    	for (int k = 0; k < size_batch; k++) {
    		tmp += dev_dZ[i*size_batch+k] * dev_A[j*size_batch+k];
      }
    	dev_dW[i*W_col+j] = tmp;
    }
  }

}

// db1(20*1) is from dZ1(20*964)
// db2(2*1) is from dZ1(2*964)
void Back_db(double* dZ, double* db, int row, int col, int size_batch)

{
  int r, c;
  for(r = 0; r < row; r++) {
    double tmp = 0;
    for(c = 0; c < col; c++) {
      tmp += dZ[r*size_batch+c];
    }
    db[r*1+0] = tmp;
  }
}
    
void Back_dA1 (double* dev_W2, double* dev_dZ2, double* dev_dA1, int size_batch, int size_hidden, int size_output, int max_row, int max_col)

{  
    
  // dA1(20*964) = dZ2(2*964) * W2(2*20)
  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) { 
      double partial = 0.0;
    	for (int k = 0; k < size_output; k++) {
    		partial += dev_W2(k,i) * dev_dZ2(k,j);
      }
    	dev_dA1(i,j) = partial;
    }
  }

}


void Back_dZ1 (double* dev_dA1, double* dev_A1, double* dev_Z1, double* dev_dZ1, int size_batch, int acti_type, int max_row, int max_col)

{  

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) { 
      if(acti_type == 1){ // Sigmoid
          dev_dZ1(i, j) = dev_dA1(i, j) * dev_A1(i, j) * (1-dev_A1(i, j)); // dZ1 = dA1*A1*(1-A1)
      } 
      else if(acti_type == 2) { // ReLU
        if(dev_Z1(i, j) < 0) 
          dev_dZ1(i, j) = 0;
        else
          dev_dZ1(i, j) = dev_dA1(i, j); //dZ1 = dA1*Z1_mask
      }
    }
  }

}

void backprop(int acti_type, int block_size) { // type = 1 is Sigmoid

  // get dZ2
  Back_dZ2(A2, Y_trn, dZ2, size_train, size_train, 1, size_train);

  // get dw2
  Back_dW(A1, dZ2, dW2, size_train, size_hidden, size_output, size_hidden);


  // get db2
  Back_db(dZ2, db2, size_output, size_train, size_train);

  // get dA1
  Back_dA1(W2, dZ2, dA1, size_train, size_hidden, size_output, size_hidden, size_train);    


  // get dZ1
  Back_dZ1(dA1, A1, Z1, dZ1, size_train, acti_type, size_hidden, size_train);


  // get dW1
  Back_dW(X_trn, dZ1, dW1, size_train, size_input, size_hidden, size_input);

  // get b1
  Back_db(dZ1, db1, size_hidden, size_train, size_train);

}

void update_Wb(double* dev_dWb, double* dev_Wb, int col, double learn_rate, int max_row, int max_col)
{

  for(int i = 0; i < max_row; i++) {
    for(int j = 0; j < max_col; j++) { 
      dev_Wb[i*col+j] = dev_Wb[i*col+j] - learn_rate * dev_dWb[i*col+j];
    }
  }
  
}

void updateParameter(double learn_rate, int block_size)
{

  // update w1
  update_Wb(dW1, W1, size_input, learn_rate, size_hidden, size_input);

  // update b1
  update_Wb(db1, b1, 1, learn_rate, size_hidden, 1);

  
  // update w2
  update_Wb(dW2, W2, size_hidden, learn_rate, size_output, size_hidden);
  
  // update b2
  update_Wb(db2, b2, 1, learn_rate, size_output, 1);
 

}


void read_X(string data_path, double* array)
{  
  ifstream inFile(data_path);  
  string row;   
  int p;
  p = 0;
  string value;
  while (getline(inFile, row)){  
    stringstream col(row);    
    while (getline(col, value, ',')){
      array[p] = stod(value);      
      p++;
    }   
  }  
}


void read_Y(string data_path, int* array)
{  
  ifstream inFile(data_path);  
  string row;   
  int p;
  p = 0;
  string value;
  while (getline(inFile, row)){  
    stringstream col(row);    
    while (getline(col, value, ',')){
      array[p] = stod(value);      
      p++;
    }   
  }  
}

/* Set the value and reading data */
void read_data()
{

  X_trn = (double *) malloc(size_X_trn * sizeof(double));  // 196*964
  Y_trn = (int *)    malloc(size_Y_trn * sizeof(int));     // 1*964
  X_tst = (double *) malloc(size_X_tst * sizeof(double));  // 196*414
  Y_tst = (int *)    malloc(size_Y_tst * sizeof(int));     // 1*414
  
  
  string X_trn_path = "X_trn.csv"; // Defined the name of cvs file
  string Y_trn_path = "Y_trn.csv";
  string X_tst_path = "X_tst.csv";
  string Y_tst_path = "Y_tst.csv";
        
  read_X(X_trn_path, X_trn); //Execution 
  read_Y(Y_trn_path, Y_trn);  
  read_X(X_tst_path, X_tst);  
  read_Y(Y_tst_path, Y_tst);  

}

void initialize_Wb() {
  
  W1 = (double *) malloc(size_W1*sizeof(double));   // 20*196
  b1 = (double *) malloc(size_b1*sizeof(double));   // 20*1
  W2 = (double *) malloc(size_W2*sizeof(double));   // 2*20
  b2 = (double *) malloc(size_b2*sizeof(double));   // 2*1
  
  dW1 = (double *) malloc(size_dW1*sizeof(double)); // 20*196
  db1 = (double *) malloc(size_db1*sizeof(double)); // 20*1
  dW2 = (double *) malloc(size_dW2*sizeof(double)); // 2*20
  db2 = (double *) malloc(size_db2*sizeof(double)); // 2*1

  memset (W1,0.5,size_W1);
  memset (b1,0,  size_b1);
  memset (W2,0.5,size_W2);
  memset (b2,0,  size_b2);
  
  memset (dW1,0, size_dW1);
  memset (db1,0, size_db1);
  memset (dW2,0, size_dW2);
  memset (db2,0, size_db2);
  
	default_random_engine e;
	uniform_real_distribution<double> u(-1,1);
 
  for (int i = 0; i < size_W1; i++) {
    W1[i] = u(e);
  }  
  for (int i = 0; i < size_W2; i++) {
    W2[i] = u(e);
  }   
  for (int i = 0; i < size_b1; i++) {
    b1[i] = 0;
  } 
  for (int i = 0; i < size_b2; i++) {
    b2[i] = 0;
  } 
  
}

double accuracy(int* max_index, int* Y, int size_batch) 
{
  
  int i;
  double count = 0;
  for(i = 0; i < size_batch; i++) {
    if(Y(0, i) == max_index(0, i))
      count += 1;
  }  
  return count/double(size_batch);
  
}

double train(double* X_trn, int* Y_trn, int acti_type, int block_size) {

  forward(X_trn, Y_trn, "train", acti_type, block_size);
  backprop(acti_type, block_size); // 1 Sigmoid 2 ReLU 
  updateParameter(0.01, block_size);
  return cross_entropy_loss(Y_trn, A2, size_train);
  
}

double test(double* X, int* Y, string type, int acti_type, int block_size) {

  forward(X, Y, type, acti_type, block_size);
  if(type == "train")
    return accuracy(max_index, Y, size_train);
  else
    return accuracy(max_index, Y, size_test);
  
}

int main(int argc, char *argv[])
{

  int block_size;
  double loss;
  double acc_trn, acc_tst;
  int e;
  int epochs = 20000;
  int acti_type = 1;
   
  if ( argc < 3 ){
    printf(" Usage: first argument: block size \n");
    printf(" second argument: activation type \n");
    return -1;
  } else if ( argc > 3 ) {
    printf("\n Too many arguments. \n");
    return -1;
  } else {
    block_size = atoi(argv[1]);
    acti_type = atoi(argv[2]);
  }
  

  initialize_Wb();
  read_data();
  float elapsed_time = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(e = 0; e < epochs; e++) {
    loss = train(X_trn, Y_trn, acti_type, block_size);
    // printf("%f \n", loss);
    // printf("the %d epoch, the training loss is: %f \n", e, loss);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf( "Elapsed Time: %.4e msec. \n", elapsed_time );
  
  acc_trn = test(X_trn, Y_trn, "train", acti_type, block_size);
  acc_tst = test(X_tst, Y_tst, "test", acti_type, block_size);
  printf("the %d epoch, the training accuracy is: %f, the test accuracy is: %f\n", e, acc_trn, acc_tst);
  
}

